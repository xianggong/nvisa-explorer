// includes, system
#include <stdlib.h>

// includes CUDA
#include <hip/hip_runtime.h>

__global__ void
testKernel(float *g_idata, float *g_odata)
{
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    hipMalloc((void **) &d_idata, mem_size);
    // copy host memory to device
    hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

    // allocate device memory for result
    float *d_odata;
    hipMalloc((void **) &d_odata, mem_size);

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads, hipMemcpyDeviceToHost);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

    exit(EXIT_SUCCESS);
}

